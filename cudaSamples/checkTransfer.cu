
#include <hip/hip_runtime.h>
// nvcc profilerExample.cu -o profileExample
int main()
{
    const unsigned int X=1048576; //1 Megabyte
    const unsigned int bytes = X*sizeof(int);
    int *hostArray= (int*)malloc(bytes);
    int *deviceArray;
    hipMalloc((int**)&deviceArray,bytes);
    memset(hostArray,0,bytes);
    hipMemcpy(deviceArray,hostArray,bytes,hipMemcpyHostToDevice);
    hipMemcpy(hostArray,deviceArray,bytes,hipMemcpyDeviceToHost);

    hipFree(deviceArray);

}
